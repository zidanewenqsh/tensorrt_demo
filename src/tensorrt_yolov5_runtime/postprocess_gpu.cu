#include "hip/hip_runtime.h"
#include "postprocess_gpu.cuh"
static const char* cocolabels[] = {
    "person", "bicycle", "car", "motorcycle", "airplane",
    "bus", "train", "truck", "boat", "traffic light", "fire hydrant",
    "stop sign", "parking meter", "bench", "bird", "cat", "dog", "horse",
    "sheep", "cow", "elephant", "bear", "zebra", "giraffe", "backpack",
    "umbrella", "handbag", "tie", "suitcase", "frisbee", "skis",
    "snowboard", "sports ball", "kite", "baseball bat", "baseball glove",
    "skateboard", "surfboard", "tennis racket", "bottle", "wine glass",
    "cup", "fork", "knife", "spoon", "bowl", "banana", "apple", "sandwich",
    "orange", "broccoli", "carrot", "hot dog", "pizza", "donut", "cake",
    "chair", "couch", "potted plant", "bed", "dining table", "toilet", "tv",
    "laptop", "mouse", "remote", "keyboard", "cell phone", "microwave",
    "oven", "toaster", "sink", "refrigerator", "book", "clock", "vase",
    "scissors", "teddy bear", "hair drier", "toothbrush"
};

static std::vector<cv::Scalar> colors = {
    cv::Scalar(255, 0, 0),      // 蓝色
    cv::Scalar(0, 255, 0),      // 绿色
    cv::Scalar(0, 0, 255),      // 红色
    cv::Scalar(0, 255, 255),    // 黄色
    cv::Scalar(255, 0, 255),    // 洋红色（品红）
    cv::Scalar(255, 255, 0),    // 青色
    cv::Scalar(0, 165, 255),    // 橙色
    cv::Scalar(128, 0, 128),    // 紫色
    cv::Scalar(255, 192, 203),  // 粉色
    cv::Scalar(128, 128, 128)   // 灰色
};

__global__ void filter_boxes_kernel(float* boxes, gBox *filtered_boxes, float* d_matrix, int num_boxes, int num_probs, 
    float conf_threshold, float nms_threshold, int *box_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_boxes) return ;

    float* box = boxes + idx * num_probs;
    float prob = box[4]; // 置信度

    if (prob < conf_threshold) return;

    // 找到最大类别概率
    float max_class_prob = 0.0f;
    int label = -1;
    for (int i = 5; i < num_probs; ++i) {
        if (box[i] > max_class_prob) {
            max_class_prob = box[i];
            label = i - 5;
        }
    }

    prob *= max_class_prob;
    if (prob < conf_threshold) return;

    int count = atomicAdd(box_count, 1);
    gBox *bboxes = filtered_boxes + count;
    // 计算边界框坐标
    float x1 = box[0] - box[2] / 2;
    float y1 = box[1] - box[3] / 2;
    float x2 = box[0] + box[2] / 2;
    float y2 = box[1] + box[3] / 2;
#if 1
    x1 = d_matrix[0] * x1 + d_matrix[1] * y1 + d_matrix[2];
    y1 = d_matrix[3] * x1 + d_matrix[4] * y1 + d_matrix[5];
    x2 = d_matrix[0] * x2 + d_matrix[1] * y2 + d_matrix[2];
    y2 = d_matrix[3] * x2 + d_matrix[4] * y2 + d_matrix[5];
#endif
    bboxes->x1 = x1;
    bboxes->y1 = y1;
    bboxes->x2 = x2;
    bboxes->y2 = y2;
    bboxes->prob = prob;
    bboxes->label = label;
    __syncthreads();
    for (int i = 0; i < count - 1; i++) {
        gBox *bbox = filtered_boxes + i;
        if (bboxes->remove || bbox->remove || bbox->label != label) continue;
        float xx1 = max(bbox->x1, x1);
        float yy1 = max(bbox->y1, y1);
        float xx2 = min(bbox->x2, x2);
        float yy2 = min(bbox->y2, y2);
        float w = max(0.0f, xx2 - xx1 + 1);
        float h = max(0.0f, yy2 - yy1 + 1);
        float inter = w * h;
        if (inter == 0) continue;
        float ovr = inter / ((bbox->x2 - bbox->x1 + 1) * (bbox->y2 - bbox->y1 + 1) + (x2 - x1 + 1) * (y2 - y1 + 1) - inter);
        if (ovr < nms_threshold) continue;
        if (prob > bbox->prob) {
            bbox->remove = 1;
        } else {
            bboxes->remove = 1;
        }
    }
}

void postprocess_cuda(float* d_data, gBox *d_filtered_boxes, int *d_box_count, float *d_matrix,  int output_numbox, int output_numprob, float confidence_threshold, float nms_threshold) {
    // int output_numel = output_numbox * output_numprob;
    // 调用 Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_numbox + threadsPerBlock - 1) / threadsPerBlock;
    filter_boxes_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_filtered_boxes, d_matrix, 
        output_numbox, output_numprob, confidence_threshold, nms_threshold, d_box_count);
    // 检查是否有错误发生
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Postprocess CUDA Kernel Error: " << hipGetErrorString(error) << std::endl;
    }
    
    // 同步设备以确保所有操作都已完成
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "Postprocess CUDA Error: " << hipGetErrorString(error) << std::endl;
    }
    // printf("d_box_count: %d\n", *d_box_count);
}


cv::Mat draw_g(gBox *boxes, int count, cv::Mat img) {
    // cv::Mat img_draw = img.clone();
    for (int i = 0; i < count; i++) {
        auto box = boxes[i];
        if (box.remove == 1) continue;
        printf("i: %d\n", i);
        box.print();
        int x1 = (int)box.x1;
        int y1 = (int)box.y1;
        int x2 = (int)box.x2;
        int y2 = (int)box.y2;
        auto name = cocolabels[box.label];
        auto caption = cv::format("%s %.2f", name, box.prob);
        auto color = colors[box.label % 10];
        cv::rectangle(img, cv::Point(x1, y1), cv::Point(x2, y2), color, 2);
        cv::putText(img, caption, cv::Point(x1, y1), cv::FONT_HERSHEY_SIMPLEX, 1, color, 2);
    }
    return img;
}